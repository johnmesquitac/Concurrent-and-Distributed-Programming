
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T  0.01
#define kappa 0.000045
#define N 100000 
#define MAX_THREAD 64

__global__ void add(double *u, double *u_prev){
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index < N)
    u[index] = u_prev[index] + kappa*dt/(dx*dx)*(u_prev[index - 1] - 2 * u_prev[index] + u_prev[index + 1]);
}

int main(void) {
  double *tmp, *u, *u_prev;
  double *a, *b;
  double x, t;
  long int i, maxloc;
  int size = N * sizeof(double);
  float temp_mili = 0.0;
  hipEvent_t start, stop;
  const int blocks = (N + MAX_THREAD -1)/MAX_THREAD;

  /* Alocando vetores */
  u = (double *) malloc((N + 1)*sizeof(double));
  u_prev = (double *) malloc((N + 1)*sizeof(double));
  hipMalloc((void **)&a, size);
  hipMalloc((void **)&b, size);

  printf("Inicio: qtde=%ld, dt=%g, dx=%g, dx²=%g, kappa=%f, const=%f\n",
         (N + 1), dt, dx, dx*dx, kappa, kappa*dt/(dx*dx));
  printf("Iteracoes previstas: %g\n", T/dt);

  hipEventCreate(&start);
  hipEventCreate(&stop);

  x = 0;
  for (i = 0; i < N + 1; i++) {
    if (x <= 0.5) u_prev[i] = 200*x;
    else        u_prev[i] = 200*(1.-x);
    x += dx;
  }

  printf("dx=%g, x=%g, x-dx=%g\n", dx, x, x-dx);
  printf("u_prev[0,1]=%g, %g\n", u_prev[0], u_prev[1]);
  printf("u_prev[n-1,n]=%g, %g\n", u_prev[N-1], u_prev[N]);

  hipEventRecord(start, 0);

  t = 0.;
  while (t<T) {
    x = dx;
    hipMemcpy(a, u, size, hipMemcpyHostToDevice);
    hipMemcpy(b, u_prev, size, hipMemcpyHostToDevice);
    add<<<blocks,MAX_THREAD>>>(a, b);
    hipMemcpy(u, a, size, hipMemcpyDeviceToHost);
    u[0] = u[N] = 0.; /* forca condicao de contorno */
    tmp = u_prev; u_prev = u; u = tmp; /* troca entre ponteiros */
    t += dt;
  }

  /* Calculando o maior valor e sua localizacao */
  maxloc = 0;
  for (i=1; i < N+1; i++) {
    if (u[i] > u[maxloc]) maxloc = i;
  }

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&temp_mili, start, stop);

  printf("Maior valor u[%ld] = %g\n", maxloc, u[maxloc]);

  printf("Tempo = %f\n", temp_mili);

  hipEventDestroy(start);
        hipEventDestroy(stop);

  free(u);
  free(u_prev);
  hipFree(a);
  hipFree(b);

  return 0;
}



